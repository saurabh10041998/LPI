
#include <hip/hip_runtime.h>
#include<iostream>
#include<time.h>
#include<climits>
#include<stdlib.h>
using namespace std;

__global__ void minimum(int *a, int *b, int n) {
    int tid = threadIdx.x;
    int min_limit = INT_MAX;
    for(int i=tid; i<min(tid+256, n); i++) {
        if(min_limit > a[i])
            min_limit = a[i];
    }
    b[tid] = min_limit;
}

int main(){
    int n = 128;
    int *a = (int *)malloc(n * sizeof(int));
    srand(time(0));
    for(int i=0;i<n;i++) {
        a[i] = rand();
    }

    for(int i=0; i<n; i++){
        cout<<a[i]<<" ";
    }
    cout<<endl;

    //creating the device array
    int *dev_a, *dev_b;
    int size = n * sizeof(int);
    float total_elapsed_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);    
    hipMalloc(&dev_a, size);
    hipMalloc(&dev_b, sizeof(int));
    int * ans;
    ans = (int *)malloc(sizeof(int));
    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipEventRecord(start, 0);
    minimum<<<1, n>>>(dev_a, dev_b, n);
    hipDeviceSynchronize();    
    hipMemcpy(ans, dev_b, sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventElapsedTime(&total_elapsed_time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    cout<<"Minimum of the array is "<<ans[0]<<endl;   
    cout<<"total elapsed time "<<total_elapsed_time<<"ms"<<endl;

    //calculating the serial way
    clock_t  start_cpu = clock();
    int min  = INT_MAX;
    for(int i=0;i<n;i++) {
        if(a[i] < min)
            min = a[i];
    }
    clock_t stop_cpu = clock();
    cout<<"Min by CPU "<<min<<endl;
    clock_t total_time = (stop_cpu - start_cpu) * 1000 / CLOCKS_PER_SEC;
    cout<<total_time<<endl;
    return 0;    
}
