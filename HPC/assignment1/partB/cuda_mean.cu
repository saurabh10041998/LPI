
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<time.h>
#include<iostream>
#define N 256*256
using namespace std;


__global__ void reduce(int *input, int *output) {
    __shared__ int shared_data[256];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    shared_data[threadIdx.x] = input[i];
    __syncthreads();
    for(int s = 1; s < blockDim.x; s*=2) {
        int index = 2 * s * threadIdx.x;
        if(index < blockDim.x){
            shared_data[index] += shared_data[index + s];
        }
        __syncthreads();
    }

    if(threadIdx.x == 0)
        atomicAdd(output, shared_data[0]);
    
}

int main() {

    int *hostA, *hostB, *dev_A, *dev_B;


    hostA = (int *)malloc(N * sizeof(int));

    hostB = (int *)malloc(N * sizeof(int));

    hipMalloc(&dev_A, N*sizeof(int));
    hipMalloc(&dev_B, N*sizeof(int));

    //initiallize host arrays
    for(int i=0;i<N;i++) {
        hostA[i] = i;
    }

    hostB[0] = 0;

    //copy on device..
    hipMemcpy(dev_A, hostA, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, hostB, N*sizeof(int), hipMemcpyHostToDevice);

    reduce<<<256, 256>>>(dev_A, dev_B);

    hipMemcpy(hostB, dev_B, N*sizeof(int), hipMemcpyDeviceToHost);
    cout<<(hostB[0]/N)<<endl;  
    return 0;     


}
